//
// Created by oke on 17.01.19.
//

#include "SortBuffer.h"
#include "cuda/cuda_utils.h"
#include "SortedBucketContainer.h"
#include "PreScan.h"

SortBuffer::SortBuffer(Scenario_id &scenario, size_t preSumBatchSize) {

    this->preSumBatchSize = preSumBatchSize;

    gpuErrchk(hipMalloc((void **) &bucketSizes, sizeof(size_t) * scenario.lanes.size()));

    lanePreSumBufferSize = GetRequiredPreSumReqBufferSize(scenario.lanes.size(), preSumBatchSize);
    gpuErrchk(hipMalloc((void **) &laneBucketPreSumBuffer, sizeof(size_t) * lanePreSumBufferSize));

    gpuErrchk(hipMalloc((void **) &pBucketData, scenario.lanes.size() * sizeof(BucketData)));
    gpuErrchk(hipMalloc((void **) &pBucketData2, scenario.lanes.size() * sizeof(BucketData)));
    gpuErrchk(hipMalloc((void **) &pBucketDataNumFilled, sizeof(unsigned int)));
    gpuErrchk(hipMalloc((void **) &pBucketDataNumFilled2, sizeof(unsigned int)));

    laneCounterSize = scenario.lanes.size();
    gpuErrchk(hipMalloc((void **) &laneCounter, laneCounterSize * sizeof(unsigned int)));

    reinsert_buffer_size = scenario.lanes.size();
    gpuErrchk(hipMalloc((void **) &reinsert_buffer, reinsert_buffer_size * sizeof(TrafficObject_id *)));

    size_t buffer_size = SortedBucketContainer::getBufferSize(scenario, 4.);
    preSumInLen = buffer_size;
    preSumOutLen = GetRequiredPreSumReqBufferSize(buffer_size, batch_count);
    assert(IsPowerOfTwo(PRE_SUM_BLOCK_SIZE));

    gpuErrchk(hipMalloc((void **) &preSumIn, preSumInLen * sizeof(size_t)));
    gpuErrchk(hipMalloc((void **) &preSumOut, preSumOutLen * sizeof(size_t)));

}

SortBuffer::~SortBuffer() {

    hipFree(preSumIn);
    hipFree(preSumOut);

    hipFree(reinsert_buffer);

    hipFree(laneCounter);

    hipFree(pBucketDataNumFilled);
    hipFree(pBucketDataNumFilled2);
    hipFree(pBucketData);
    hipFree(pBucketData2);

    hipFree(laneBucketPreSumBuffer);

    hipFree(bucketSizes);

}