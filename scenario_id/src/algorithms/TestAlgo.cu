#include "hip/hip_runtime.h"
//
// Created by oke on 09.01.19.
//

#include <hiprand/hiprand_kernel.h>
#include <model/Lane.h>
#include "algorithms/TestAlgo.h"
#include "cuda/cuda_utils.h"


struct BucketData {
    size_t size;
    size_t buffer_size;
    TrafficObject_id **buffer;
};

class BucketMemory {
public:
    size_t bucket_count;
    BucketData *buckets;
    TrafficObject_id **main_buffer;
    size_t main_buffer_size;


    CUDA_HOSTDEV BucketMemory() : bucket_count(0), buckets(nullptr), main_buffer(nullptr), main_buffer_size(0) {

    }

    CUDA_HOSTDEV static size_t getBufferSize(CudaScenario_id *scenario, float bucket_memory_factor) {
        size_t total_buffer_size = 0;
        for (Lane_id &l : scenario->getLaneIterator()) {
            total_buffer_size += ceil(bucket_memory_factor * scenario->getRoad(l.road)->length / 5.);
        }
        return total_buffer_size;
    }

    CUDA_HOSTDEV BucketMemory(CudaScenario_id *scenario, BucketData *_buckets, TrafficObject_id **_main_buffer, float bucket_memory_factor) {
        bucket_count = scenario->getNumLanes();
        buckets = _buckets; // new BucketData[scenario->getNumLanes()];
        assert(buckets != nullptr);

        size_t total_buffer_size = 0;
        size_t i = 0;
        for (Lane_id &l : scenario->getLaneIterator()) {
            buckets[i].size = 0;
            buckets[i].buffer_size = ceil(bucket_memory_factor * scenario->getRoad(l.road)->length / 5.);
            total_buffer_size += buckets[i].buffer_size;
            i++;
        }

        main_buffer = _main_buffer; // new TrafficObject_id*[total_buffer_size];
        assert(main_buffer != nullptr);
        this->main_buffer_size = total_buffer_size;

        size_t offset = 0;
        for(i = 0; i < scenario->getNumLanes(); i++) {
            buckets[i].buffer = main_buffer + offset;
            offset += buckets[i].buffer_size;
        }
        printf("Allocated: %.2fMB\n", (float) (sizeof(size_t) * scenario->getNumLanes() * 2 + sizeof(TrafficObject_id**) * scenario->getNumLanes() +
                                               sizeof(TrafficObject_id*) * total_buffer_size) / 1024. / 1024.);
    }

    CUDA_HOSTDEV ~BucketMemory() {
        //if (buckets != nullptr) free(buckets);
        //if (main_buffer != nullptr) free(main_buffer);
    }



};


template<> template<>
BucketContainer<TrafficObject_id*, nullptr> BucketContainer<TrafficObject_id*, nullptr>::construct<CudaScenario_id>(CudaScenario_id &scenario) {
    BucketContainer<TrafficObject_id *, nullptr> bucketContainer(scenario.getNumLanes(), 15);
    for(auto &c : scenario.getCarIterator()) {
        auto &bucket = bucketContainer[c.lane];
        bucket.resize(bucket.getSize() + 1);
        bucket[bucket.getSize() - 1] = &c;
    }
    assert(bucketContainer.numElements() == scenario.getNumCars());
    return bucketContainer;

}

__device__
size_t GetGlobalIdx(){
    return + blockIdx.z * blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y
           + blockIdx.y * blockDim.x * blockDim.y * blockDim.z * gridDim.x
           + blockIdx.x * blockDim.x * blockDim.y * blockDim.z
           + threadIdx.z * blockDim.y * blockDim.x
           + threadIdx.y * blockDim.x
           + threadIdx.x;
}
__device__
size_t GetThreadIdx(){
    return threadIdx.z * blockDim.y * blockDim.x
           + threadIdx.y * blockDim.x
           + threadIdx.x;
}
__device__
size_t GetBlockIdx(){
    return blockIdx.z * gridDim.y * gridDim.x
           + blockIdx.y * gridDim.x
           + blockIdx.x;
}

__device__
size_t GetGridDim(){
    return gridDim.y * gridDim.x * gridDim.z;
}

__device__
size_t GetBlockDim(){
    return blockDim.y * blockDim.x * blockDim.z;
}


__global__ void allocateKernel(CudaScenario_id *device_scenario, BucketContainer<TrafficObject_id *, nullptr> *device_bucketContainer) {
    size_t lane_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (lane_idx == 0) *device_bucketContainer = BucketContainer<TrafficObject_id *, nullptr>(device_scenario->getNumLanes(), 0);
}

__global__ void constructKernel(CudaScenario_id *device_scenario, BucketContainer<TrafficObject_id *, nullptr> *device_bucketContainer) {
    size_t lane_idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (lane_idx >= device_bucketContainer->getNumBuckets()) return;

    auto &bucket = (*device_bucketContainer)[lane_idx];

    for(auto &c : device_scenario->getCarIterator()) {
        if(lane_idx == c.lane) {
            bucket.resize(50);
            continue;
            bucket.resize(bucket.getSize() + 1);
            bucket[bucket.getSize() - 1] = &c;
        }
    }
    //printf("%lu, %lu ---- %lu/%lu: %lu\n", (size_t )threadIdx.x, (size_t )blockIdx.x, lane_idx, device_bucketContainer->getNumBuckets(), bucket.getSize());

}
__global__ void checkKernel(CudaScenario_id *device_scenario, BucketContainer<TrafficObject_id *, nullptr> *device_bucketContainer) {
    assert(device_bucketContainer->numElements() == device_scenario->getNumCars());
}

template<> template<>
void BucketContainer<TrafficObject_id*, nullptr>::construct_device<CudaScenario_id>(CudaScenario_id *device_scenario, BucketContainer<TrafficObject_id *, nullptr> *device_bucketContainer, size_t num_lanes) {

    unsigned int BLOCK_NUM = ceil((float) num_lanes / THREAD_NUM);
    dim3 threads(THREAD_NUM, 1);
    dim3 blocks(BLOCK_NUM, 1);
    allocateKernel<<<1, 1>>>(device_scenario, device_bucketContainer);
    constructKernel<<<blocks, threads>>>(device_scenario, device_bucketContainer);
#ifdef DEBUG_MSGS
    checkKernel<<<1, 1>>>(device_scenario, device_bucketContainer);
#endif
}


template<typename T>
__device__ void cuda_swap(T &t1, T &t2) {
    T t = t1;
    t1 = t2;
    t2 = t;
}


template<typename T>
__global__ void bitonic_sort_step_kernel(T *dev_values, int j, int k, int n) {
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = GetGlobalIdx();
    ixj = i ^ j;

    if (i >= n || ixj >= n)
        return;
    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj]) {
                /* exchange(i,ixj); */
                cuda_swap(dev_values[i], dev_values[ixj]);
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj]) {
                /* exchange(i,ixj); */
                cuda_swap(dev_values[i], dev_values[ixj]);
            }
        }
    }
}


template<typename T>
__global__ void bitonic_sort_merge_kernel(T* values, int k, int n) {
    unsigned int i; /* Sorting partners: i and ixj */
    i = GetGlobalIdx();
    if(i + k < n && values[i] > values[i + k])
        cuda_swap(values[i], values[i + k]);
}

#define THREADS 512 // 2^9
template <typename T>
void dev_mem_bitonic_sort(T *device_values, unsigned long n) {
    unsigned long block_num = (unsigned int) ceil(n / (float) THREADS);
    unsigned long block_num2 = 1;
    // printf("%d Threads on %lux%lu Blocks\n", THREADS, block_num, block_num2);
    if (block_num > 65535) {
        block_num2 = 65535;
        block_num = (int) ceil((float) block_num / (float) block_num2);
    }
    dim3 blocks(block_num, block_num2);    /* Number of blocks   */
    dim3 threads(THREADS, 1);  /* Number of threads  */
    int j, k;
    /* Major step */
    for (k = 2; k <= n; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            bitonic_sort_step_kernel<<<blocks, threads>>>(device_values, j, k, n);
            gpuErrchk( hipPeekAtLastError() );
        }
    }

    unsigned long power = pow(2, floor(log(n)/log(2)));
    for (unsigned long k = power; k > 0; k >>= 1) {
        bitonic_sort_merge_kernel<<<blocks, threads>>>(device_values, k, n);
        gpuErrchk( hipPeekAtLastError() );
    }
}

template<typename T, typename Cmp>
__device__ void bitonic_sort_step(T *dev_values, int j, int k, int n, Cmp cmp) {
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * threadIdx.y;
    ixj = i ^ j;

    if (i >= n || ixj >= n)
        return;
    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (cmp(dev_values[ixj], dev_values[i])) {
                /* exchange(i,ixj); */
                cuda_swap(dev_values[i], dev_values[ixj]);
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (cmp(dev_values[i], dev_values[ixj])) {
                /* exchange(i,ixj); */
                cuda_swap(dev_values[i], dev_values[ixj]);
            }
        }
    }
}


template<typename T, typename Cmp>
__device__ void bitonic_sort_merge(T* values, int k, int n, Cmp cmp) {
    unsigned int i; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * threadIdx.y;
    if(i + k < n && cmp(values[i + k], values[i]))
        cuda_swap(values[i], values[i + k]);
}

template <typename Cmp>
__global__ void cudaSortKernel(BucketMemory *container, Cmp cmp) {
    size_t first_bucket_idx = blockIdx.x + gridDim.x * blockIdx.y;
    size_t i = threadIdx.x + blockDim.x * threadIdx.y;
    size_t block_count = gridDim.x * gridDim.y;
    size_t thread_count = blockDim.x * blockDim.y;
    size_t buckets_per_thread = ceilf((float) container->bucket_count / block_count);

    for(size_t bucket_idx = first_bucket_idx; bucket_idx < container->bucket_count; bucket_idx += block_count) {
#if DEBUG_MSGS
        // if (first_bucket_idx == 0 && i == 0) printf("bucket_idx: %lu - block count: %lu\n", bucket_idx, block_count);
#endif
        if (container->buckets[bucket_idx].size > 1) {
            TrafficObject_id **device_values = container->buckets[bucket_idx].buffer;
            size_t n = container->buckets[bucket_idx].size;
            // printf("bucket %lu with %lu items.\n", bucket_idx, n);
            int j, k;
            /* Major step */
            for (k = 2; k <= n; k <<= 1) {
                /* Minor step */
                for (j = k >> 1; j > 0; j = j >> 1) {
                    bitonic_sort_step(device_values, j, k, n, cmp);
                    __syncthreads();
                }
            }

            unsigned long power = pow(2., floor(log((double) n) / log(2.)));
            for (unsigned long k = power; k > 0; k >>= 1) {
                bitonic_sort_merge(device_values, k, n, cmp);
                __syncthreads();
            }
        }
    }
}
__global__ void checkSortKernel(BucketMemory *container) {
    TrafficObject_id::Cmp cmp;
    size_t bucket_id = GetGlobalIdx();

    if (bucket_id >= container->bucket_count) return;
    if(bucket_id == 0) printf("Testing Sorting of Container...\n");
    if (container->buckets[bucket_id].size == 0) return;
    //printf("Lane %d\n", i);
    for(int j = 0; j < container->buckets[bucket_id].size - 1; j++) {
        //printf("%lu,%lu: %f <= %f (%d, %d)\n", buckets[i][j]->id, buckets[i][j + 1]->id,  buckets[i][j]->x, buckets[i][j + 1]->x,
        //        cmp(buckets[i][j], buckets[i][j + 1]), buckets[i][j]->x <= buckets[i][j + 1]->x);
        assert(cmp(container->buckets[bucket_id].buffer[j], container->buckets[bucket_id].buffer[j + 1]) && container->buckets[bucket_id].buffer[j]->x <= container->buckets[bucket_id].buffer[j + 1]->x);
    }
}


template <typename ObjectType, ObjectType Z, typename Cmp>
__global__ void sort_cu(BucketContainer<ObjectType, Z> *container, Cmp cmp)
{
    __shared__ int res;
    res = 0;
    size_t fid = -1;
    int bucket_id = blockIdx.y;
    auto &bucket = (*container)[bucket_id];

    int i = blockIdx.x;
    int j = threadIdx.x;

    if(bucket_id == fid && j == 0 && i == 0) printf("index(%d-%d) size: %lu\n", i, j, bucket.getSize());
    if(i >= bucket.getSize())
        return;


    __shared__ ObjectType array[1024];

    if(j < bucket.getSize())
        array[j] = bucket[j];

    __syncthreads();

    if(j < bucket.getSize()) {
        if (bucket_id == fid) printf("index(%d-%d) %p/%p\n", i, j, array[i], array[j]);
        //if (bucket_id == fid) printf("%p: %p\n", array + j, array[j]);
        if (cmp(array[j], array[i]) || (i > j && !cmp(array[i], array[j]))) {
            // if ((array[i] > array[j]) || (i > j && array[i] == array[j])) {
            atomicAdd((unsigned int *) &res, 1);
            if(bucket_id == fid) printf("index(%d-%d): %f/%f - %d\n", i, j, array[i]->x, array[j]->x , res);
        }
    }

    __syncthreads();

    if(bucket_id == fid && j == 0) printf("index(%d): %d\n", i, res);
    if(j < bucket.getSize())
        bucket[res] = array[i];
}

template <typename ObjectType, ObjectType Z>
template<typename Cmp>
void BucketContainer<ObjectType, Z>::sort_device_bucket(BucketContainer<ObjectType, Z> *container, Cmp cmp, size_t num_buckets) {

    /*dim3 blocks(1024, num_buckets);
    dim3 threads(50, 1);
    sort_cu<<<blocks, threads>>>(container, cmp);
    gpuErrchk( hipPeekAtLastError() );*/
    dim3 blocks(MIN(2048, num_buckets), 1);    /* Number of blocks   */
    dim3 threads(20, 1);  /* Number of threads  */
//    cudaSortKernel<<<blocks, threads>>>(container, cmp);

}




__global__ void find_nearest(CudaScenario_id *scenario, BucketMemory *container, TrafficObject_id **nearest_left,
        TrafficObject_id **nearest_own, TrafficObject_id **nearest_right) {
    AlgorithmWrapper wrapper(*scenario);
#ifdef DEBUG_MSGS
    size_t print_for_id = -1;
#endif
    size_t car_idx = GetGlobalIdx();
    int lane_offset = (int)(car_idx % 3) - 1;
    car_idx /= 3;

    if (car_idx >= scenario->getNumCars())
        return;

    TrafficObject_id car = *scenario->getCar(car_idx);
    size_t lane_id = (size_t ) -1;
    TrafficObject_id **nearest = nullptr;
    Road_id::NeighboringLanes n_lanes;
    switch (lane_offset) {
        case 1:
            n_lanes = wrapper.getNeighboringLanes(*scenario->getLane(car.lane));
            lane_id = n_lanes.right;
            nearest = nearest_right;
            break;
        case 0:
            lane_id = car.lane;
            nearest = nearest_own;
            break;
        case -1:
            n_lanes = wrapper.getNeighboringLanes(*scenario->getLane(car.lane));
            lane_id = n_lanes.left;
            nearest = nearest_left;
            break;
        default:
            assert(false);
    }
    car.lane = lane_id;
    size_t n = container->buckets[lane_id].size;
    TrafficObject_id *&nearest_font = nearest[car_idx];
    TrafficObject_id *&nearest_back = nearest[car_idx + scenario->getNumCars()];

    if (lane_id == (size_t) -1) {
        nearest_back = nullptr;
        nearest_font = nullptr;
        return;
    }
    if (n == 0) {
        nearest_back = nullptr;
        nearest_font = nullptr;
    } else {

        TrafficObject_id **lane_objects = container->buckets[lane_id].buffer;
        size_t search_idx = n / 2;
        size_t from = 0;
        size_t to = n;

        if (n == 1) {
            if (*lane_objects[0] == car) {
                nearest_font = nullptr;
                nearest_back = nullptr;
            } else if (*lane_objects[0] > car) {
                nearest_font = lane_objects[0];
                nearest_back = nullptr;
            } else if (*lane_objects[0] < car) {
                nearest_back = lane_objects[0];
                nearest_font = nullptr;
            }
        } else {
            while (true) {
#ifdef DEBUG_MSGS
                if (car.id == print_for_id)
                    printf("Find(%lu): %lu/%.2f, Current(%lu) %lu/%.2f, Index: %lu/%lu/%lu \n", car.id, car.lane, car.x,
                           lane_objects[search_idx]->id, lane_objects[search_idx]->lane, lane_objects[search_idx]->x,
                           from,
                           search_idx, to);
#endif
                if (*lane_objects[search_idx] < car) {
                    if (search_idx + 1 == n || *lane_objects[search_idx + 1] >= car) {
                        break;
                    }
                    from = search_idx + 1;
                    search_idx += (to - from) / 4 == 0 ? 1 : (to - from) / 4;
                } else {
                    to = search_idx;
                    search_idx -= (to - from) / 4 == 0 ? 1 : (to - from) / 4;
                }
                if ((to - from) == 1)
                    break;
            }

#ifdef DEBUG_MSGS
            if (car.id == print_for_id)
                printf("Find(%lu): %lu/%.2f, Current(%lu) %lu/%.2f, Index: %lu/%lu/%lu \n", car.id, car.lane, car.x,
                       lane_objects[search_idx]->id, lane_objects[search_idx]->lane, lane_objects[search_idx]->x, from,
                       search_idx, to);
#endif

            assert(search_idx < n && (*lane_objects[search_idx] < car || search_idx == 0));

            if (search_idx == 0 && *lane_objects[search_idx] >= car) {
                nearest_back = nullptr;
                while (search_idx < n && *lane_objects[search_idx] == car) search_idx++;
                if (search_idx < n)
                    nearest_font = lane_objects[search_idx];
            } else {
                if (search_idx < n)
                    nearest_back = lane_objects[search_idx];
                search_idx++;
                while (search_idx < n && *lane_objects[search_idx] == car) search_idx++;
                if (search_idx < n)
                    nearest_font = lane_objects[search_idx];
            }
        }
    }

#ifdef DEBUG_MSGS
    if(car.id == print_for_id) {
        printf("Found(%lu): %lu %lu\n", car.id,
               (nearest_back != nullptr ? nearest_back->id : (size_t) -1),
               nearest_font != nullptr ? nearest_font->id : (size_t) -1);
    }
#endif

    Lane_id *l = scenario->getLane(car.lane);
    RedTrafficLight_id *tl = scenario->getLight(l->traffic_light);
    if(tl->isRed()) {
        if (car < *tl && (nearest_font == nullptr || *tl < *nearest_font)) {
            nearest_font = tl;
        }
        if (car > *tl && (nearest_back == nullptr || *tl > *nearest_back)) {
            nearest_back = tl;
        }
    }


}

__device__ void test_right_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    printf("test_right_lane_neighbors\n");
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if (car_id >= scenario->getNumLanes()) return;

    Road_id::NeighboringLanes lanes = algorithmWrapper.getNeighboringLanes(*scenario->getLane(scenario->getCar(car_id)->lane));
    if(lanes.right == (size_t) -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
        assert(neighbors[car_id] == nullptr);
        return;
    }
    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(lanes.right));

    if (neig.back == -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front == -1) {
        if(neighbors[car_id] != nullptr) printf("%lu\n", neighbors[car_id]->id);

        assert(neighbors[car_id] == nullptr);
    } else {
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__device__ void test_left_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    printf("test_left_lane_neighbors\n");
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if (car_id >= scenario->getNumLanes()) return;

    Road_id::NeighboringLanes lanes = algorithmWrapper.getNeighboringLanes(*scenario->getLane(scenario->getCar(car_id)->lane));
    if(lanes.left == (size_t) -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
        assert(neighbors[car_id] == nullptr);
        return;
    }
    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(lanes.left));

    if (neig.back == -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front == -1) {
        assert(neighbors[car_id] == nullptr);
    } else {
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__device__ void test_own_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    printf("test_own_lane_neighbors\n");
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if (car_id >= scenario->getNumLanes()) return;

    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(scenario->getCar(car_id)->lane));

    if (neig.back == -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front == -1) {
        assert(neighbors[car_id] == nullptr);
    } else {
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__global__ void test_neighborsKernel(CudaScenario_id *scenario, TrafficObject_id ** dev_left_neighbors,TrafficObject_id ** dev_own_neighbors,TrafficObject_id ** dev_right_neighbors) {
    test_right_lane_neighbors(dev_right_neighbors, scenario);
    test_own_lane_neighbors(dev_own_neighbors, scenario);
    test_left_lane_neighbors(dev_left_neighbors, scenario);
}

__global__
void bucketMemoryInitializeKernel(BucketMemory *bucketmem,  BucketData *buckets, TrafficObject_id **main_buffer, CudaScenario_id *cuda_device_scenario, float bucket_memory_factor) {
    new(bucketmem)BucketMemory(cuda_device_scenario, buckets, main_buffer, bucket_memory_factor);
}

__global__
void bucketMemoryLoadKernel(BucketMemory *bucketmem, CudaScenario_id *cuda_device_scenario, float bucket_memory_factor) {
    size_t lane_idx = GetGlobalIdx();

    if (lane_idx >= bucketmem->bucket_count) return;

    BucketData &bucket = bucketmem->buckets[lane_idx];

    for(auto &c : cuda_device_scenario->getCarIterator()) {
        if(lane_idx == c.lane) {
            assert(bucket.size + 1 < bucket.buffer_size);
            bucket.size += 1;
            bucket.buffer[bucket.size - 1] = &c;
        }
    }
    //printf("%lu, %lu ---- %lu/%lu: %lu\n", (size_t )threadIdx.x, (size_t )blockIdx.x, lane_idx, device_bucketContainer->getNumBuckets(), bucket.getSize());
}

__global__ void kernel_get_changes(Car_id::AdvanceData *change, CudaScenario_id * scenario_data,
                                   TrafficObject_id **right_lane_neighbors, TrafficObject_id **own_lane_neighbors, TrafficObject_id **left_lane_neighbors) {
    size_t car_idx = threadIdx.x + blockDim.x * blockIdx.x + blockDim.y * blockIdx.y;
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    if (car_idx < scenario_data->getNumCars()) {
        Lane_id::NeighboringObjectsRef rightNeighbors(right_lane_neighbors[car_idx], right_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        Lane_id::NeighboringObjectsRef ownNeighbors(own_lane_neighbors[car_idx], own_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        Lane_id::NeighboringObjectsRef leftNeighbors(left_lane_neighbors[car_idx], left_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        change[car_idx] = algorithm_wrapper.nextStep(*scenario_data->getCar(car_idx), leftNeighbors, ownNeighbors, rightNeighbors);
    }
}


__global__ void updateSignalsKernel(CudaScenario_id * scenario_data) {
    size_t jnt_idx = threadIdx.x + blockDim.x * blockIdx.x + blockDim.y * blockIdx.y;
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    if (jnt_idx < scenario_data->getNumJunctions()) {
        algorithm_wrapper.updateSignals(*scenario_data->getJunction(jnt_idx));
    }
}


__global__ void testChangesKernel(CudaScenario_id *scenario_data, Car_id::AdvanceData *device_changes) {
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    size_t car_idx = GetGlobalIdx();
    if(car_idx == 0) printf("testKernelChanges\n");
    if (car_idx >= scenario_data->getNumCars()) return;
    Car_id::AdvanceData change = algorithm_wrapper.nextStep(*scenario_data->getCar(car_idx));
    assert(change.car == device_changes[car_idx].car);
    assert(change.lane_offset == device_changes[car_idx].lane_offset);
    assert(change.acceleration == device_changes[car_idx].acceleration);
}


__global__ void applyChangesKernel(Car_id::AdvanceData *change, CudaScenario_id * scenario_data) {
    size_t car_idx = threadIdx.x + blockDim.x * blockIdx.x + blockDim.y * blockIdx.y;
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    if (car_idx < scenario_data->getNumCars()) {
        algorithm_wrapper.advanceStep(*scenario_data->getCar(change[car_idx].car), change[car_idx]);
    }
}

__global__ void testBucketsForInvalidLaneKernel(BucketMemory *container) {
    size_t bucket_id = GetGlobalIdx();
    if (bucket_id >= container->bucket_count) return;
    if(bucket_id == 0) printf("testing container buckets\n");
    if (container->buckets[bucket_id].size == 0) return;
    //printf("Lane %d\n", i);
    for(int j = 0; j < container->buckets[bucket_id].size - 1; j++) {
        assert(container->buckets[bucket_id].buffer[j]->lane == bucket_id);
    }
}

__device__ inline bool isInWrongLane(BucketMemory *container, TrafficObject_id **object) {
    if (*object == nullptr) return false;
    BucketData supposed_bucket = container->buckets[(*object)->lane];
    return (object < supposed_bucket.buffer || supposed_bucket.buffer + supposed_bucket.size <= object);
}

CUDA_HOSTDEV inline bool IsPowerOfTwo(unsigned long x)
{
    return (x != 0) && ((x & (x - 1)) == 0);
}


__device__ void PreScan(size_t *temp, size_t idx, size_t n, size_t skip=1) {

    assert(IsPowerOfTwo(n));
    assert(IsPowerOfTwo(skip));
    if (!(2 * idx < n)) printf("%lu, %lu\n", 2*idx, n);
    assert(2 * idx < n);

    int offset = 1;

    n /= skip;

    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();

        if (idx < d) {
            int ai = offset * (2 * idx + 1) - 1;
            int bi = offset * (2 * idx + 2) - 1;

            temp[bi] += temp[ai * skip];
        }
        offset *= 2;
    }
    size_t total_sum = 0;
    if (idx == 0) {
        total_sum = temp[n - 1];
        temp[n - 1] = 0;
    } // clear the last element

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();

        if (idx < d) {
            int ai = offset * (2 * idx + 1) - 1;
            int bi = offset * (2 * idx + 2) - 1;
            //printf("%d, %d, %d, %d\n", ai, bi, offset, idx);

            size_t t = temp[ai * skip];
            temp[ai * skip] = temp[bi * skip];
            temp[bi * skip] += t;
        }
    }

    __syncthreads();
    size_t t1 = temp[2 * idx];
    size_t t2 = temp[2 * idx + 1];
    __syncthreads();

    if (idx != 0)
        temp[2 * idx - 1] = t1;
    else
        temp[n - 1] = total_sum;
    temp[2 * idx] = t2;
}
__global__ void BlockWisePreScan(BucketMemory *container, size_t *g_odata, size_t n) {
    assert(IsPowerOfTwo(n));

    __syncthreads();
    extern __shared__ size_t temp[];// allocated on invocation

    size_t traffic_object_id = GetThreadIdx();
    size_t buffer_offset = GetBlockIdx() * n;


    TrafficObject_id **p_obj1 = container->main_buffer + 2 * traffic_object_id + buffer_offset;
    TrafficObject_id **p_obj2 = container->main_buffer + 2 * traffic_object_id + 1 + buffer_offset;
    // if(GetGlobalIdx() == 0) printf("Starting...\n");

    if(p_obj1 < container->main_buffer + container->main_buffer_size && p_obj1 != nullptr) {
        temp[2 * traffic_object_id] = isInWrongLane(container, p_obj1) ? 1 : 0; // load input into shared memory
        //printf("Got one. %lu, %lu, %lu \n", traffic_object_id, buffer_offset, temp[2 * traffic_object_id]);
    } else
        temp[2 * traffic_object_id] = 0;

    if(p_obj2 < container->main_buffer + container->main_buffer_size && p_obj2 != nullptr) {
        temp[2 * traffic_object_id + 1] = isInWrongLane(container, p_obj2) ? 1 : 0;  // load input into shared memory
        //printf("Got one. %lu, %lu, %lu \n", traffic_object_id, buffer_offset, temp[2 * traffic_object_id + 1]);
    } else
        temp[2 * traffic_object_id + 1] = 0;

    PreScan(temp, traffic_object_id, n);

    __syncthreads();

    if(p_obj1 < container->main_buffer + container->main_buffer_size)
        g_odata[2 * traffic_object_id + buffer_offset] = temp[2*traffic_object_id]; // write results to device memory
    if(p_obj2 < container->main_buffer + container->main_buffer_size)
        g_odata[2 * traffic_object_id+1 + buffer_offset] = temp[2*traffic_object_id+1];

}

__global__ void collect_changedKernel(BucketMemory *container) {
    __shared__ unsigned long int count;
    BucketData last_bucket = container->buckets[container->bucket_count - 1];
    TrafficObject_id **it = container->main_buffer + GetGlobalIdx();
    if(it < last_bucket.buffer + last_bucket.size) {
        if ((*it) == nullptr) return;
        BucketData supposed_bucket = container->buckets[(**it).lane];
        if(it < supposed_bucket.buffer || supposed_bucket.buffer + supposed_bucket.size <= it) {
            printf("%lu is in wrong lane (actual: %lu). (%lu) \n", (**it).id, (**it).lane, (it - container->main_buffer));
        }
    }

}

__global__ void MergeBlockWisePreScan(size_t *out, size_t *in, size_t n, size_t buffer_size, size_t skip_count) {
    assert(IsPowerOfTwo(n));

    size_t idx = GetThreadIdx();
    size_t offset = GetBlockIdx() * n * skip_count;

    extern __shared__ size_t temp[];
    if ((idx + 1) * skip_count - 1 < buffer_size)
        temp[2 * idx] = in[(2 * idx + 1) * skip_count - 1 + offset];
    else
        temp[2 * idx] = 0;

    if ((idx + 1) * skip_count - 1 < buffer_size)
        temp[2 * idx + 1] = in[(2 * idx + 2) * skip_count - 1 + offset];
    else
        temp[2 * idx + 1] = 0;

    __syncthreads();

    PreScan(temp, idx, n);

    __syncthreads();

    out[2 * idx + n * GetBlockIdx()] = temp[2 * idx];
    out[2 * idx + 1 + n * GetBlockIdx()] = temp[2 * idx + 1];
}

__global__ void MergeBlockWisePreScanStep2(size_t *out, size_t *in, size_t n, size_t buffer_size, size_t out_size) {
    size_t idx = GetThreadIdx();
    size_t offset = GetBlockIdx() * n;

    if (GetBlockIdx() > 0 && idx + offset < out_size) {
        out[idx + offset] += in[GetBlockIdx() - 1];
    }

}

__global__ void GetIndicesKernel(BucketMemory *container, size_t *prefixSum, size_t n,
        TrafficObject_id **reinsert_buffer, size_t buffer_size) {
    size_t idx = GetGlobalIdx();
    if (idx >= n) return;
    // if(idx == 256) printf("%lu - %lu\n" , prefixSum[255], prefixSum[256]);
    if ((idx == 0 && prefixSum[0] > 0) || (idx != 0 && prefixSum[idx] != prefixSum[idx - 1])){
        // printf("%lu\n", container->main_buffer_size);
        size_t insert_id = idx == 0 ? 0 : prefixSum[idx - 1];
        reinsert_buffer[insert_id] = container->main_buffer[idx];
        container->main_buffer[idx] = nullptr;
        /*printf("Buffer(%lu)_chng: %lu -> %lu, car(%lu)\n", idx, idx == 0 ? 0 : prefixSum[idx - 1], prefixSum[idx],
                container->main_buffer[idx] == nullptr ? -1 : container->main_buffer[idx]->id);*/

        // container->buckets[container->main_buffer[idx - 1]->lane][prefixSum[idx - 1]]
    }

    __syncthreads();
    //printf("%lu,%lu - %lu,%lu,%lu|%lu,%lu,%lu\n", prefixSum[100800], prefixSum[100801],
    //        prefixSum[98301], prefixSum[98302], prefixSum[98303], prefixSum[98304], prefixSum[98305], prefixSum[98306]);
}

__global__ void InsertChanged(BucketMemory *container, size_t bufferSize,
        size_t *prefixSum, size_t prefixSumLength, TrafficObject_id **reinsertBuffer, size_t reinsertBufferLength) {

    assert(IsPowerOfTwo(bufferSize));

    size_t idx = GetThreadIdx();
    size_t lane_id = GetBlockIdx();

    extern __shared__ size_t insert_into_lane[];


    if (2 * idx + 1 >= bufferSize) return;
    if (prefixSum[prefixSumLength - 1] == 0) return;
    if(2 * idx < prefixSum[prefixSumLength - 1])
        insert_into_lane[2 * idx] = (size_t) (reinsertBuffer[2 * idx]->lane == lane_id);
    else
        insert_into_lane[2 * idx] = 0;


    if(2 * idx + 1 < prefixSum[prefixSumLength - 1])
        insert_into_lane[2 * idx + 1] = (size_t) (reinsertBuffer[2 * idx + 1]->lane == lane_id);
    else
        insert_into_lane[2 * idx + 1] = 0;

    PreScan(insert_into_lane, idx, bufferSize);

    __syncthreads();

    // printf("Insert %lu new Cars.\n", insert_into_lane[bufferSize - 1]);

    auto &bucket = container->buckets[lane_id];
    if (idx >= prefixSum[prefixSumLength - 1]) return;
    if ((idx == 0 && insert_into_lane[0] > 0) || (idx != 0 && insert_into_lane[idx] != insert_into_lane[idx - 1])) {
        size_t insert_id = idx == 0 ? 0 : insert_into_lane[idx - 1];
        assert(bucket.buffer[bucket.size + insert_id] == nullptr);
        //printf("lane: %lu, bucket idx: %lu\n", lane_id, insert_id + bucket.size);
        bucket.buffer[bucket.size + insert_id] = reinsertBuffer[idx];
    }

    __syncthreads();

    if(idx == 0) bucket.buffer_size += insert_into_lane[bufferSize - 1];

}

__global__ void FixSizeKernel(BucketMemory *container) {

    size_t idx = GetThreadIdx();
    size_t lane_id = GetBlockIdx();

    auto &bucket = container->buckets[lane_id];

    __shared__ size_t new_size;
    if (idx >= bucket.buffer_size) return;
    if (idx == 0) {
        if (bucket.buffer[0] == nullptr)
            new_size = 0;
    } else {
        if (bucket.buffer[idx] == nullptr && bucket.buffer[idx - 1] != nullptr)
            new_size = idx;
    }

    __syncthreads();

    if (idx == 0) {
       // printf("%lu: %lu\n", lane_id, new_size);
        bucket.size = new_size;
    }
}

#define CHECK_FOR_ERROR() hipDeviceSynchronize(); gpuErrchk( hipPeekAtLastError() );

void collect_changed(CudaScenario_id *scenario, BucketMemory *container) {

#ifdef RUN_WITH_TESTS
    collect_changedKernel<<<1024,1024>>>(container);
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
#endif

    size_t buffer_size = BucketMemory::getBufferSize(scenario, 4.);
    size_t block_size = 512;
    assert(IsPowerOfTwo(block_size));

    std::vector<size_t> preSumHost(buffer_size);

    size_t *preSum;
    gpuErrchk(hipMalloc((void**) &preSum, buffer_size * sizeof(size_t)));

    BlockWisePreScan<<<ceil((float) buffer_size / block_size), block_size / 2, block_size * sizeof(size_t)>>>(container, preSum, block_size);
    CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
#endif

    std::vector<size_t *> reduce_arrays;
    reduce_arrays.push_back(preSum);
    std::vector<size_t> reduce_sizes;
    reduce_sizes.push_back(buffer_size);

    size_t initial_temp_size = ceil((float) buffer_size / block_size);
    size_t temp_size = initial_temp_size;
    size_t block_size_i;
    for(block_size_i=block_size; block_size_i < buffer_size; block_size_i *= block_size) {

        size_t *temp;
        gpuErrchk(hipMalloc((void**) &temp, temp_size * sizeof(size_t) ));

        MergeBlockWisePreScan<<<ceil((float) temp_size / block_size), block_size / 2., block_size * sizeof(size_t )>>>(temp, reduce_arrays.back(), block_size, reduce_sizes.back(), block_size);

        // printf("%f\n",ceil((float) temp_size / block_size) );
        CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk(hipMemcpy(preSumHost.data(), temp, temp_size * sizeof(size_t), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
#endif
        reduce_arrays.push_back(temp);
        reduce_sizes.push_back(temp_size);

        temp_size = ceil((float)temp_size / block_size);
    }
    while(reduce_arrays.size() != 1) {
        size_t *temp = reduce_arrays.back();
        reduce_arrays.pop_back();
        temp_size = reduce_sizes.back();
        reduce_sizes.pop_back();

#ifdef RUN_WITH_TESTS
        gpuErrchk(hipMemcpy(preSumHost.data(), temp, temp_size * sizeof(size_t), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
#endif

        MergeBlockWisePreScanStep2<<<temp_size, block_size, block_size * sizeof(size_t )>>>(reduce_arrays.back(), temp, block_size, reduce_sizes.back(), reduce_sizes.back());
        CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
#endif

        gpuErrchk( hipFree(temp) );
    }


#ifdef RUN_WITH_TESTS
    gpuErrchk(hipMemcpy(preSumHost.data(), reduce_arrays.back(), reduce_sizes.back() * sizeof(size_t), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
#endif

    TrafficObject_id **reinsert_buffer;
    gpuErrchk(hipMalloc((void**) &reinsert_buffer, 400 * sizeof(TrafficObject_id*)));

    GetIndicesKernel<<<buffer_size / 256 + 1, 256>>>(container, preSum, buffer_size, reinsert_buffer, 400.);
    CHECK_FOR_ERROR()

    InsertChanged<<<scenario->getNumLanes(), 256, 512 * sizeof(size_t )>>>(container, 512, preSum, buffer_size, reinsert_buffer, 400.);
    CHECK_FOR_ERROR()

    dim3 blocks(MIN(2048, scenario->getNumLanes()), 1);    /* Number of blocks   */
    dim3 threads(20, 1);  /* Number of threads  */

    TrafficObject_id::Cmp cmp;
    cudaSortKernel<<<blocks, threads>>>(container, cmp);
    CHECK_FOR_ERROR();

    FixSizeKernel<<<scenario->getNumLanes(), 1024>>>(container);
    CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
    TrafficObject_id *bla[buffer_size];
    BucketMemory mem;
    gpuErrchk(hipMemcpy(&mem, container, sizeof(BucketMemory), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(bla, mem.main_buffer, buffer_size * sizeof(TrafficObject_id*), hipMemcpyDeviceToHost));


    gpuErrchk(hipMemcpy(preSumHost.data(), preSum, buffer_size * sizeof(size_t), hipMemcpyDeviceToHost));
    // hipDeviceSynchronize();

    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
    printf("collect_changed---\n");
#endif

    gpuErrchk(hipFree(preSum));
}

__global__ void prescank(size_t *tmp) {
    PreScan(tmp, GetThreadIdx(), GetBlockDim() * 2, 1);
}

void TestAlgo::advance(size_t steps) {
/*
    constexpr size_t arr_size = 16;
    size_t ceil_arr_size = pow(2, ceil(log(arr_size)/log(2)));
    size_t my_array[arr_size];
    size_t prefix_sum[arr_size];
    size_t i;
    for (i = 0; i < arr_size; i++) {
        my_array[i] = rand() > std::numeric_limits<int>::max() / 2 ? 1 : 0;
    }

    size_t *dev_my_array;
    gpuErrchk(hipMalloc((void**) &dev_my_array, ceil_arr_size * sizeof(size_t)));
    size_t *dev_my_array_sorted;
    gpuErrchk(hipMalloc((void**) &dev_my_array_sorted, ceil_arr_size * sizeof(size_t)));


    gpuErrchk(hipMemcpy(dev_my_array, my_array, ceil_arr_size * sizeof(size_t), hipMemcpyHostToDevice));

    //dev_mem_bitonic_sort(dev_my_array, arr_size);
    prescank<<<1, ceil_arr_size/2>>>(dev_my_array);

    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk(hipMemcpy(prefix_sum, dev_my_array, arr_size * sizeof(size_t), hipMemcpyDeviceToHost));

    size_t offset = 0;
    for(int i = 0; i < arr_size; i++) {
        offset += my_array[i];
        printf("%lu, %lu, %lu\n", prefix_sum[i], offset, my_array[i]);
        //assert(prefix_sum[i] == offset);
    }
    printf("\n");
    exit(0);*/
    /*
    constexpr size_t arr_size = 1024;
    int my_array[arr_size];
    int i;
    for (i = 0; i < arr_size; i++) {
        my_array[i] = rand();
    }

    int *dev_my_array;
    gpuErrchk(hipMalloc((void**) &dev_my_array, arr_size * sizeof(int)));
    int *dev_my_array_sorted;
    gpuErrchk(hipMalloc((void**) &dev_my_array_sorted, arr_size * sizeof(int)));

    gpuErrchk(hipMemcpy(dev_my_array, my_array, arr_size * sizeof(int), hipMemcpyHostToDevice));

    //dev_mem_bitonic_sort(dev_my_array, arr_size);
    sort_cu<<<arr_size, arr_size>>>(dev_my_array, dev_my_array_sorted);
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk(hipMemcpy(my_array, dev_my_array_sorted, arr_size * sizeof(int), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    //std::cout << my_array[0] << std::endl;
    for (i = 1; i < arr_size; i++) {
        //std::cout << my_array[i] << std::endl;
        if(my_array[i - 1] > my_array[i]) {
            std::cout << "ERROR." << std::endl;
            exit(0);
        }
    }*/
    Scenario_id *__scenario = getIDScenario();
    CudaScenario_id scenario = CudaScenario_id::fromScenarioData(*__scenario);

    CudaScenario_id *device_cuda_scenario = CudaScenario_id::fromScenarioData_device(*__scenario);

    TrafficObject_id **main_buffer;
    gpuErrchk(hipMalloc((void**) &main_buffer, BucketMemory::getBufferSize(&scenario, 4) * sizeof(TrafficObject_id*)));

    BucketData *buckets;
    gpuErrchk(hipMalloc((void**) &buckets, scenario.getNumLanes() * sizeof(BucketData)));

    // allocate bucket class
    BucketMemory *bucket_memory;
    gpuErrchk(hipMalloc((void**) &bucket_memory, sizeof(BucketMemory)));

    // initialize bucket class
    bucketMemoryInitializeKernel<<<1, 1>>>(bucket_memory, buckets, main_buffer, device_cuda_scenario, 4.);
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );

    // load data into memory
    bucketMemoryLoadKernel<<<1024, 1024>>>(bucket_memory, device_cuda_scenario, 4.);
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );

    TrafficObject_id **dev_left_neighbors, **dev_own_neighbors, **dev_right_neighbors;
    gpuErrchk(hipMalloc((void**) &dev_left_neighbors, 2 * scenario.getNumCars() * sizeof(TrafficObject_id*)));
    gpuErrchk(hipMalloc((void**) &dev_own_neighbors, 2 * scenario.getNumCars() * sizeof(TrafficObject_id*)));
    gpuErrchk(hipMalloc((void**) &dev_right_neighbors, 2 * scenario.getNumCars() * sizeof(TrafficObject_id*)));


    TrafficObject_id::Cmp cmp;

    dim3 blocks(MIN(2048, scenario.getNumLanes()), 1);    /* Number of blocks   */
    dim3 threads(20, 1);  /* Number of threads  */

    Car_id::AdvanceData *device_changes;
    gpuErrchk(hipMalloc((void**) &device_changes, getIDScenario()->cars.size() * sizeof(Car_id::AdvanceData)));

/*
    dim3 blocks(ceil(sqrt(scenario.getNumCars())), ceil(sqrt(scenario.getNumCars())));
    dim3 threads(1, 3);
*/
    // test_neighbors(device_bucketContainer, device_cuda_scenario, &scenario);
#ifdef DEBUG_MSGS
    printf("start steps\n");
#endif

    for(int i = 0; i < steps; i++) {
#ifdef DEBUG_MSGS
        printf("Step: %d\n", i);
#endif
        /*BucketContainer<TrafficObject_id *, nullptr>::sort_device_bucket(device_bucketContainer, cmp, scenario.getNumLanes());
        gpuErrchk( hipPeekAtLastError() );*/

        cudaSortKernel<<<blocks, threads>>>(bucket_memory, cmp);
        CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        checkSortKernel<<<scenario.getNumLanes(), 1>>>(bucket_memory);

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
#endif

        find_nearest<<<blocks, threads>>>(device_cuda_scenario, bucket_memory, dev_left_neighbors, dev_own_neighbors, dev_right_neighbors);
        CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        test_neighborsKernel<<<1, 1>>>(device_cuda_scenario, dev_left_neighbors, dev_own_neighbors, dev_right_neighbors);

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
#endif

        kernel_get_changes<<<512, 512>>>(device_changes, device_cuda_scenario, dev_right_neighbors, dev_own_neighbors, dev_left_neighbors);
        CHECK_FOR_ERROR()

#ifdef RUN_WITH_TESTS

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        testChangesKernel<<<512, 512>>>(device_cuda_scenario, device_changes);

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

#endif
        applyChangesKernel<<<512, 512>>>(device_changes, device_cuda_scenario);
        CHECK_FOR_ERROR()

        collect_changed(&scenario, bucket_memory);

        // prescan<<<bucket_memory->bucket_count, ceil_arr_size/2, ceil_arr_size * sizeof(size_t)>>>(dev_my_array_sorted, dev_my_array, arr_size, ceil_arr_size);

#ifdef RUN_WITH_TESTS

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        testBucketsForInvalidLaneKernel<<<512, 512>>>(bucket_memory);

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

#endif
        updateSignalsKernel<<<512, 512>>>(device_cuda_scenario);
        gpuErrchk( hipPeekAtLastError() );
    }

    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );

    device_cuda_scenario->retriveData(getIDScenario());

/*    gpuErrchk(hipFree(device_changes));
    gpuErrchk( hipPeekAtLastError() );*/

};