#include "hip/hip_runtime.h"
//
// Created by oke on 09.01.19.
//

#include <hiprand/hiprand_kernel.h>
#include <model/Lane_id.h>
#include "algorithms/CudaAlgorithm.h"
#include "cuda_utils/cuda_utils.h"
#include <chrono>
#include <thread>

#include "cuda_utils/SortedBucketContainer.h"
#include "cuda_utils/PreScan.h"
#include "cuda_utils/SortBuffer.h"

__device__ void test_right_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if(car_id == 0) printf("test_right_lane_neighbors\n");
    if (car_id >= scenario->getNumCars()) return;


    Road_id::NeighboringLanes lanes = algorithmWrapper.getNeighboringLanes(*scenario->getLane(scenario->getCar(car_id)->lane));
    if(lanes.right == (size_t) -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
        assert(neighbors[car_id] == nullptr);
        return;
    }
    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(lanes.right));

    if (neig.back == (size_t )-1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front ==  (size_t )-1) {
        if(neighbors[car_id] != nullptr) printf("%lu\n", neighbors[car_id]->id);

        assert(neighbors[car_id] == nullptr);
    } else {

        if(!(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front)) {
            printf("Car(%lu, %lu): %lu == %lu, tllane(%lu)\n", car_id, scenario->getCar(car_id)->lane, neighbors[car_id] != nullptr ? neighbors[car_id]->id : (size_t ) -1, neig.front, scenario->getLight(neig.front)->lane);
        }
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__device__ void test_left_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if(car_id == 0) printf("test_left_lane_neighbors\n");
    if (car_id >= scenario->getNumCars()) return;


    Road_id::NeighboringLanes lanes = algorithmWrapper.getNeighboringLanes(*scenario->getLane(scenario->getCar(car_id)->lane));
    if(lanes.left == (size_t) -1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
        assert(neighbors[car_id] == nullptr);
        return;
    }
    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(lanes.left));

    if (neig.back == (size_t )-1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front ==(size_t ) -1) {
        assert(neighbors[car_id] == nullptr);
    } else {
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__device__ void test_own_lane_neighbors(TrafficObject_id **neighbors, CudaScenario_id *scenario) {
    AlgorithmWrapper algorithmWrapper(*scenario);
    size_t car_id = GetGlobalIdx();
    if(car_id == 0) printf("test_own_lane_neighbors\n");

    if (car_id >= scenario->getNumCars()) return;

    Lane_id::NeighboringObjects neig = algorithmWrapper.getNeighboringObjects(*scenario->getCar(car_id), *scenario->getLane(scenario->getCar(car_id)->lane));

    if (neig.back == (size_t )-1) {
        assert(neighbors[scenario->getNumCars() + car_id] == nullptr);
    } else {
        if(!(neighbors[scenario->getNumCars() +car_id] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back)) {
            printf("c - %5lu, FrontCar(%5lu/%5lu) at Lane(%lu)\n", car_id, neig.back, scenario->getNumCars(), scenario->getCar(neig.back)->lane);
        }
        assert(neighbors[scenario->getNumCars() +car_id ] != nullptr && neighbors[scenario->getNumCars() +car_id]->id == neig.back);
    }

    if (neig.front == (size_t )-1) {
        assert(neighbors[car_id] == nullptr);
    } else {
        assert(neighbors[car_id] != nullptr && neighbors[car_id]->id == neig.front);
    }
}

__global__ void test_neighborsKernel(CudaScenario_id *scenario, TrafficObject_id ** dev_left_neighbors,TrafficObject_id ** dev_own_neighbors,TrafficObject_id ** dev_right_neighbors) {
    test_right_lane_neighbors(dev_right_neighbors, scenario);
    test_own_lane_neighbors(dev_own_neighbors, scenario);
    test_left_lane_neighbors(dev_left_neighbors, scenario);
}

__global__ void kernel_get_changes(Car_id::AdvanceData *change, CudaScenario_id * scenario_data,
                                   TrafficObject_id **right_lane_neighbors, TrafficObject_id **own_lane_neighbors, TrafficObject_id **left_lane_neighbors) {
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    CUDA_GLOBAL_ITER(car_idx, scenario_data->getNumCars()) {
        Lane_id::NeighboringObjectsRef rightNeighbors(right_lane_neighbors[car_idx], right_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        Lane_id::NeighboringObjectsRef ownNeighbors(own_lane_neighbors[car_idx], own_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        Lane_id::NeighboringObjectsRef leftNeighbors(left_lane_neighbors[car_idx], left_lane_neighbors[car_idx + scenario_data->getNumCars()]);
        change[car_idx] = algorithm_wrapper.nextStep(*scenario_data->getCar(car_idx), leftNeighbors, ownNeighbors, rightNeighbors);
#ifdef RUN_WITH_TESTS
        if(car_idx == CAR_TO_ANALYZE) {
            printf("Changes for car(%lu) with Neighbors l(%lu, %lu), o(%lu, %lu), r(%lu, %lu): (%d, %f)\n", car_idx,
                   left_lane_neighbors[car_idx + scenario_data->getNumCars()] == nullptr ? (size_t )-1 : left_lane_neighbors[car_idx + scenario_data->getNumCars()]->id,
                   left_lane_neighbors[car_idx] == nullptr ? (size_t )-1 : left_lane_neighbors[car_idx]->id,
                   own_lane_neighbors[car_idx + scenario_data->getNumCars()] == nullptr ? (size_t )-1 : own_lane_neighbors[car_idx + scenario_data->getNumCars()]->id,
                   own_lane_neighbors[car_idx] == nullptr ? (size_t )-1 : own_lane_neighbors[car_idx]->id,
                   right_lane_neighbors[car_idx + scenario_data->getNumCars()] == nullptr ? (size_t )-1 : right_lane_neighbors[car_idx + scenario_data->getNumCars()]->id,
                   right_lane_neighbors[car_idx] == nullptr ? (size_t )-1 : right_lane_neighbors[car_idx]->id,
                   change[car_idx].lane_offset, change[car_idx].acceleration);
        }
#endif
    }
}


__global__ void updateSignalsKernel(CudaScenario_id * scenario_data) {
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    CUDA_GLOBAL_ITER(jnt_idx, scenario_data->getNumJunctions()) {
        algorithm_wrapper.updateSignals(*scenario_data->getJunction(jnt_idx));
    }
}


__global__ void testChangesKernel(CudaScenario_id *scenario_data, Car_id::AdvanceData *device_changes) {
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    for(size_t car_idx = GetGlobalIdx(); car_idx < scenario_data->getNumCars(); car_idx += GetGlobalDim()) {
        if (car_idx == 0) printf("testKernelChanges\n");
        Car_id::AdvanceData change = algorithm_wrapper.nextStep(*scenario_data->getCar(car_idx));
        if (!(change.lane_offset == device_changes[car_idx].lane_offset &&
              change.acceleration == device_changes[car_idx].acceleration)) {
            printf("Wrong change on lane(%7lu) - expected: (%5lu, %d, %.2f) got: (%lu, %d, %.2f)\n",
                   scenario_data->getCar(change.car)->lane, change.car, change.lane_offset,
                   change.acceleration, device_changes[car_idx].car, device_changes[car_idx].lane_offset,
                   device_changes[car_idx].acceleration);

        }
        assert(change.car == device_changes[car_idx].car);
        assert(change.lane_offset == device_changes[car_idx].lane_offset);
        assert(change.acceleration == device_changes[car_idx].acceleration);
    }
}


__global__ void applyChangesKernel(Car_id::AdvanceData *change, CudaScenario_id * scenario_data) {
    AlgorithmWrapper algorithm_wrapper(*scenario_data);
    CUDA_GLOBAL_ITER(car_idx, scenario_data->getNumCars()) {
        algorithm_wrapper.advanceStep(*scenario_data->getCar(change[car_idx].car), change[car_idx]);
    }
}

__global__ void testBucketsForInvalidLaneKernel(SortedBucketContainer *container, CudaScenario_id *scenario) {
    size_t bucket_id = GetBlockIdx();
    size_t object_idx = GetThreadIdx();
    TrafficObject_id::Cmp cmp;

    if(bucket_id == 0 && object_idx == 0) printf("container validity check\n");

    if (bucket_id < container->bucket_count) {
        auto &bucket = container->buckets[bucket_id];
        if (object_idx < bucket.size) {
            TrafficObject_id *object = bucket.buffer[object_idx];
            if(object == nullptr) printf("Wrong length in %lu\n", bucket_id);
            assert(object != nullptr);
            if(object->lane != bucket_id) {
                printf("Car(%lu) is not in Bucket(%lu)\n", object->id, object->lane);
                printf("Bucket(%lu) contents: ", container->buckets[object->lane].id);
                for (int i = 0; i < container->buckets[object->lane].size; i++) {
                    TrafficObject_id *p_obj = container->buckets[object->lane].buffer[i];
                    printf(" %lu(%.2f), ", p_obj == nullptr ? (size_t )-1 : p_obj->id, p_obj == nullptr ? -1. : p_obj->x);
                }
                printf("\n");
                printf("Bucket(%lu) contents: ", bucket.id);
                for (int i = 0; i < bucket.size; i++) {
                    TrafficObject_id *p_obj = bucket.buffer[i];
                    printf(" %lu(%.2f), ", p_obj == nullptr ? (size_t )-1 : p_obj->id, p_obj == nullptr ? -1. : p_obj->x);
                }
                printf("\n");
            }
            assert(object->lane == bucket_id);
            if (object_idx > 0) {
                if (!cmp(bucket.buffer[object_idx - 1], object)) {
                    printf("Sorting error in Lane(%lu)\n", bucket_id);
                }
                assert(cmp(bucket.buffer[object_idx - 1], object));
            }
        }
    }

    if (scenario != nullptr) {
        size_t car_idx = GetGlobalIdx();
        if (car_idx < scenario->getNumCars()) {
            auto &c = *scenario->getCar(car_idx);
            assert(c.lane < scenario->getNumLanes());
            auto &b = container->buckets[c.lane];
            bool found = false;
            for (int i = 0; i < b.size; i++) {
                assert(b.buffer[i] != nullptr);
                if (b.buffer[i]->id == c.id) {
                    found = true;
                    break;
                }
            }
            if (!found) printf("Car(%lu) not in container\n", car_idx);
            assert(found);
        }
    }
}

__device__ void find_nearest_for_car_on_lane(CudaScenario_id *scenario, SortedBucketContainer *container, TrafficObject_id &car, TrafficObject_id *&front, TrafficObject_id *&back) {

    BucketData &bucket = container->buckets[car.lane];
    TrafficObject_id::Cmp cmp;
    TrafficObject_id **traffic_object = upper_bound<TrafficObject_id*>(bucket.buffer, bucket.buffer + bucket.size, &car, cmp);
    assert(traffic_object <= bucket.buffer + bucket.size && traffic_object >= bucket.buffer);
    assert(bucket.size <= bucket.buffer_size);
#ifdef DEBUG_MSGS
    if (car.id == CAR_TO_ANALYZE) printf("Lane(%lu) has %lu objects\n", car.lane, bucket.size);
#endif
    if(bucket.buffer + bucket.size != traffic_object) {
        front = *traffic_object;
#ifdef DEBUG_MSGS
        if (car.id == CAR_TO_ANALYZE) printf("Car(%lu, %lu) front: Car(%lu)\n", car.id, car.lane, (*traffic_object)->id);
#endif
    } else {
#ifdef DEBUG_MSGS
        if (car.id == CAR_TO_ANALYZE) printf("Car(%lu, %lu) has clear heading.\n", car.id, car.lane);
#endif
        front = nullptr;
    }
    do {
        traffic_object--;
    } while(traffic_object > bucket.buffer && **traffic_object >= car);

    assert(traffic_object <= bucket.buffer + bucket.size && traffic_object >= bucket.buffer - 1);

    if(traffic_object >= bucket.buffer && bucket.buffer + bucket.size != traffic_object && **traffic_object < car) {
#ifdef DEBUG_MSGS
        if (car.id == CAR_TO_ANALYZE) printf("Car(%lu, %lu) back: Car(%lu)\n",  car.id, car.lane, (*traffic_object)->id);
#endif
        back = *traffic_object;
    } else {
#ifdef DEBUG_MSGS
        if (car.id == CAR_TO_ANALYZE) printf("Car(%lu, %lu) has clear back.\n", car.id, car.lane);
#endif
        back = nullptr;
    }
}


__global__ void find_nearest2(CudaScenario_id *scenario, SortedBucketContainer *container, TrafficObject_id **nearest_left,
                              TrafficObject_id **nearest_own, TrafficObject_id **nearest_right) {

    AlgorithmWrapper wrapper(*scenario);
    CUDA_GLOBAL_ITER(idx, scenario->getNumCars() * 3) {
        int lane_offset = (int) (idx % 3) - 1;
        size_t car_idx = idx / 3;

        TrafficObject_id car = *scenario->getCar(car_idx);
        assert(scenario->getCar(car_idx) != nullptr);

        size_t lane_id = (size_t) -1;
        TrafficObject_id **nearest = nullptr;
        Road_id::NeighboringLanes n_lanes;
        switch (lane_offset) {
            case 1:
                n_lanes = wrapper.getNeighboringLanes(*scenario->getLane(car.lane));
                lane_id = n_lanes.right;
                nearest = nearest_right;
                break;
            case 0:
                lane_id = car.lane;
                nearest = nearest_own;
                break;
            case -1:
                n_lanes = wrapper.getNeighboringLanes(*scenario->getLane(car.lane));
                lane_id = n_lanes.left;
                nearest = nearest_left;
                break;
            default:
                assert(false);
        }
#ifdef DEBUG_MSGS
        if (car.id == CAR_TO_ANALYZE) printf("original lane of Car(%lu) is Lane(%lu)\n", car.id, car.lane);
#endif
        car.lane = lane_id;
        TrafficObject_id *&nearest_font = nearest[car_idx];
        TrafficObject_id *&nearest_back = nearest[car_idx + scenario->getNumCars()];

        if (lane_id == (size_t) -1) {
            nearest_back = nullptr;
            nearest_font = nullptr;
            continue;
        }

        assert(lane_id < container->bucket_count);
        if (container->buckets[lane_id].size == 0) {
            nearest_back = nullptr;
            nearest_font = nullptr;
        } else {
            find_nearest_for_car_on_lane(scenario, container, car, nearest_font, nearest_back);
        }

        Lane_id *l = scenario->getLane(car.lane);
        assert(l != nullptr);
        RedTrafficLight_id *tl = scenario->getLight(l->traffic_light);
        assert(tl != nullptr);
        if (tl->isRed()) {
            if (car < *tl && (nearest_font == nullptr || *tl < *nearest_font)) {
                nearest_font = tl;
#ifdef DEBUG_MSGS
                if (car.id == CAR_TO_ANALYZE)
                    printf("Car(%lu, %lu) has light(%lu, %lu) in the front.\n", car.id, car.lane, nearest_font->id,
                           nearest_font->lane);
#endif
            }
            /**
            if (car > *tl && (nearest_back == nullptr || *tl > *nearest_back)) {
                // nearest_back = tl;
#ifdef DEBUG_MSGS
                if (car.id == CAR_TO_ANALYZE)
                    printf("Car(%lu, %lu) has light(%lu, %lu) in the back.\n", car.id, car.lane, nearest_back->id,
                           nearest_back->lane);
#endif
            }*/
        }

    }
}

void static_advance(size_t steps, Scenario_id &scenario) {

    SortBuffer preSumBuffer(scenario, SUGGESTED_THREADS);
    size_t number_of_lanes = scenario.lanes.size();
    size_t number_of_cars = scenario.cars.size();

    CudaScenario_id *device_cuda_scenario = CudaScenario_id::fromScenarioData_device(scenario);
    std::shared_ptr<SortedBucketContainer> bucket_memory = SortedBucketContainer::fromScenario(scenario, device_cuda_scenario, preSumBuffer);

    TrafficObject_id **dev_left_neighbors, **dev_own_neighbors, **dev_right_neighbors;
    GPU_ALLOC((void **) &dev_left_neighbors, 2 * number_of_cars * sizeof(TrafficObject_id *))
    GPU_ALLOC((void **) &dev_own_neighbors, 2 * number_of_cars * sizeof(TrafficObject_id *))
    GPU_ALLOC((void **) &dev_right_neighbors, 2 * number_of_cars * sizeof(TrafficObject_id *))

    Car_id::AdvanceData *device_changes;
    GPU_ALLOC((void **) &device_changes, scenario.cars.size() * sizeof(Car_id::AdvanceData))

#ifdef DEBUG_MSGS
    printf("Starting to advance scenario...\n\n");
#endif
#ifdef RUN_WITH_TESTS
    if(CAR_TO_ANALYZE < scenario.cars.size())
        printf("Car(%lu) on Lane(%lu)\n", (size_t) CAR_TO_ANALYZE, scenario.cars[CAR_TO_ANALYZE].lane);
#endif
    for (int i = 0; i < steps; i++) {
#ifdef DEBUG_MSGS
#endif
        SortedBucketContainer::RestoreValidState(scenario, bucket_memory.get(), preSumBuffer);

#ifdef RUN_WITH_TESTS
        testBucketsForInvalidLaneKernel<<<number_of_lanes, 1024>>>(bucket_memory.get(), device_cuda_scenario);
        CHECK_FOR_ERROR();
#endif

        find_nearest2<<<scenario.cars.size() / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>(device_cuda_scenario, bucket_memory.get(), dev_left_neighbors, dev_own_neighbors, dev_right_neighbors);
        CHECK_FOR_ERROR();

#ifdef RUN_WITH_TESTS
        test_neighborsKernel<<<number_of_cars / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>
            (device_cuda_scenario, dev_left_neighbors, dev_own_neighbors, dev_right_neighbors);
        CHECK_FOR_ERROR();
#endif

        kernel_get_changes<<<number_of_cars / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>
            (device_changes, device_cuda_scenario, dev_right_neighbors, dev_own_neighbors, dev_left_neighbors);
        CHECK_FOR_ERROR();

#ifdef RUN_WITH_TESTS
        testChangesKernel<<<number_of_cars / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>
            (device_cuda_scenario, device_changes);
        CHECK_FOR_ERROR();
#endif

        applyChangesKernel<<<number_of_cars / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>(device_changes, device_cuda_scenario);
        CHECK_FOR_ERROR();

        updateSignalsKernel<<<scenario.junctions.size() / SUGGESTED_THREADS + 1, SUGGESTED_THREADS>>>(device_cuda_scenario);
        CHECK_FOR_ERROR();

    }

    device_cuda_scenario->retriveData(&scenario);

    gpuErrchk(hipFree(dev_left_neighbors));
    gpuErrchk(hipFree(dev_own_neighbors));
    gpuErrchk(hipFree(dev_right_neighbors));
    gpuErrchk(hipFree(device_changes));
}

void CudaAlgorithm::advance(size_t steps) {
    static_advance(steps, *getIDScenario());
    hipDeviceReset();
};